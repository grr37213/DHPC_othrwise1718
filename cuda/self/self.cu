#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define diff(a,b) b-a

void swap_f(float * a, float * b)
{
	float tmp = *a;
	*a = *b;
	*b = tmp;
}

float rnd_f(float min, float max)
{
	if(min > max)
	{
		swap_f(&min, &max);
	}
	return (float)rand()/(float)(RAND_MAX/diff(min,max));
}

struct body
{
	float x, y, vx, vy, m;
};

body randombody	(	
					body min, body max
				)
{
	body b =	{
					rnd_f(min.x, max.x),
					rnd_f(min.y, max.y),
					rnd_f(min.vx, max.vx),
					rnd_f(min.vy, max.vy),
					rnd_f(min.m, max.m)
				};
	return b;
}

__global__ gravitate(body* bef, body *aft, int bodycount, float * stepsize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(bodycount < (idx + 1)) return 0;

	//clear the velocity value in aft to 0
	aft[idx].vx = 0.0;
	aft[idx].vy = 0.0;

	for(int i = 0; i < blockDim.x .x; i++)
	{
		float dx = bef[i].x - bef[idx].x;
		float dy = bef[i].y - bef[idx].y;

		float sqlen = dx * dx + dy + dy;
		float acceleration = bef[idx].m / sqlen;
		//normalize
		sqlen = sqrt(sqlen);
		dx = dx/sqlen;
		dy = dx/sqlen;

		//store in new value
		//not use += operator because of multiply-add compiler optimization
		//TODO test if it means anything

		aft[idx].vx = acceleration * dx + aft[idx].vx;
		aft[idx].vy = acceleration * dy + aft[idx].vy;

		//actual new velocity here then :
		aft[idx].vx += bef[idx].vx;

		//step
		aft[idx].x = aft[idx].vx * *stepsize;
		aft[idx].y = aft[idx].vy * *stepsize;
	}
}

#define LOOP_SAFETY 10000000

int run (unsigned int bodycount, float stepsize, float simtime, unsigned int blocksize, body * hostdata)
{
	unsigned int blockamount = bodycount / blocksize +1;

	int safetycounter = LOOP_SAFETY;

	body *d_1; body *d_2;

	hipMalloc(d_1, sizeof(body) * bodycount);
	hipMalloc(d_2, sizeof(body) * bodycount);

	//initialize the masses with 0
	//this is important since when there would be an uneven/suboptimal values of bodies threads would start in vain and calculate the force of their body
	//with no mass this calculation will not influence the result

	for(int i = 0; i < bodycount ; i++)
	{
		d_1[i].m = 0.0;
		d_2[i].m = 0.0;
	}

	hipMemcpy(d_1, hostdata, bodycount*sizeof(body), hipMemcpyHostToDevice);

	unsigned int blockamount = bodycount / blocksize;
	if(bodycount%blocksize) blockamount++;

	while (simtime > 0 && safetycounter > 0)
	{
		gravitate <<<  >>>	


		safetycounter--;
	}

	free(h_1);
}

#define DEFAULT_SIMTIME 10.0
#define DEFAULT_STEPSIZE 1.0
#define DEFAULT_BODYCOUNT 1024
#define DEFAULT_BLOCKSIZE 16

void help()
{
	printf("\n Help for nbody : \n\t nbody [Number of bodys] -[flag] [[flagvalue]]* \n\n Arguments : \n");
	printf("\t none\t Number of bodys to calculate as base 10 integer (Default : %d)\n", DEFAULT_BODYCOUNT);
	printf("\t-h\t Show this help\n");
	printf("\t-s\t Stepsize of simulation. Simulation will have simtime /stepsize round trips of duration given by step and one additional trip of simtime mod stepsize (Default : %f)\n", DEFAULT_STEPSIZE);
	printf("\t-t\t Time to simulate as a Float(Default : %f)\n", DEFAULT_SIMTIME);
	printf("\t-b\t Size of a Block / amount of threads on one Block (Default : %d\n)", DEFAULT_BLOCKSIZE);
	printf("\t-m[in/ax]\t A minimal/maximal set of values for random generated bodies. Write 5 floats for x/y position , x/y velocity and mass.\nExample\n\t\t -min [minx] [miny] [minvx] [minvy] [minm]\n")
	fflush(stdout);
}


int main (int argc, char ** argv)
{
	if(argc < 1)
	{
		printf("Wrong number of arguments !\nTry \n\n\t $ nbody -h\n\nfor help !");
	}

	int argi;

	//default initializations
	unsigned int bodycount = DEFAULT_BODYCOUNT;
	float stepsize = DEFAULT_STEPSIZE;
	float simtime = DEFAULT_SIMTIME;

	unsigned int blocksize = DEFAULT_BLOCKSIZE;

	body bmin = {-10000.0, -10000.0, -100.0, -100.0, 0};
	body bmax = {10000.0, 10000.0, 100.0, 100.0, 1000};

	//argument handling 
	for(argi = 0; argi < argc; argi++)
	{
		if(argv[argi][0] == '-')
		{
			switch(argv[argi][1])
			{
				case 'h':
					help(); exit(1);
				break;
				case 't':
					simtime = atof(argv[++argi]);
				break;
				case 's':
					stepsize = atof(argv[++argi]);
				break;
				case 'b':
					blocksize = atoi(argv[++argi]);
				break;
				case 'm':
					{
						body * bchange = (argv[argi][2] == 'i')? &bmin : &bmax;
						*bchange = {atof(++argi), atof(++argi),atof(++argi),atof(++argi),atof(++argi)};
				}
				break;
			}
		}
		else
		{
			bodycount = atoi(argv[argi]);
		}
	}

	//seed random number generator
	time_t t;
	srand((unsigned) time(&t));
	//in the following a random number can be generated

	//init data
	//allocate host mem
	body* h_1 = (body*)malloc(bodycount);

	//initialize random
	{
		for (body* ptr = h_1 ; ptr < h_1+bodycount; ptr++)
		{
			*ptr = randombody(bmin, bmax);
		}
	}

	return run(bodycount, stepsize, simtime, blocksize, h_1);
}

